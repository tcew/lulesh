#include "hip/hip_runtime.h"
#include<kernels/kernelDefines.cu>

static __device__ __forceinline__
void CalcMonoGradient(Real_t *x, Real_t *y, Real_t *z,
                      Real_t *xv, Real_t *yv, Real_t *zv,
                      Real_t vol,
                      Real_t *delx_zeta,
                      Real_t *delv_zeta,
                      Real_t *delx_xi,
                      Real_t *delv_xi,
                      Real_t *delx_eta,
                      Real_t *delv_eta)
{

   #define SUM4(a,b,c,d) (a + b + c + d)
   const Real_t ptiny = Real_t(1.e-36) ;
   Real_t ax,ay,az ;
   Real_t dxv,dyv,dzv ;

   Real_t norm = Real_t(1.0) / ( vol + ptiny ) ;

   Real_t dxj = Real_t(-0.25)*(SUM4(x[0],x[1],x[5],x[4]) - SUM4(x[3],x[2],x[6],x[7])) ;
   Real_t dyj = Real_t(-0.25)*(SUM4(y[0],y[1],y[5],y[4]) - SUM4(y[3],y[2],y[6],y[7])) ;
   Real_t dzj = Real_t(-0.25)*(SUM4(z[0],z[1],z[5],z[4]) - SUM4(z[3],z[2],z[6],z[7])) ;

   Real_t dxi = Real_t( 0.25)*(SUM4(x[1],x[2],x[6],x[5]) - SUM4(x[0],x[3],x[7],x[4])) ;
   Real_t dyi = Real_t( 0.25)*(SUM4(y[1],y[2],y[6],y[5]) - SUM4(y[0],y[3],y[7],y[4])) ;
   Real_t dzi = Real_t( 0.25)*(SUM4(z[1],z[2],z[6],z[5]) - SUM4(z[0],z[3],z[7],z[4])) ;

   Real_t dxk = Real_t( 0.25)*(SUM4(x[4],x[5],x[6],x[7]) - SUM4(x[0],x[1],x[2],x[3])) ;
   Real_t dyk = Real_t( 0.25)*(SUM4(y[4],y[5],y[6],y[7]) - SUM4(y[0],y[1],y[2],y[3])) ;
   Real_t dzk = Real_t( 0.25)*(SUM4(z[4],z[5],z[6],z[7]) - SUM4(z[0],z[1],z[2],z[3])) ;

   /* find delvk and delxk ( i cross j ) */
   ax = dyi*dzj - dzi*dyj ;
   ay = dzi*dxj - dxi*dzj ;
   az = dxi*dyj - dyi*dxj ;

   *delx_zeta = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

   ax *= norm ;
   ay *= norm ;
   az *= norm ;

   dxv = Real_t(0.25)*(SUM4(xv[4],xv[5],xv[6],xv[7]) - SUM4(xv[0],xv[1],xv[2],xv[3])) ;
   dyv = Real_t(0.25)*(SUM4(yv[4],yv[5],yv[6],yv[7]) - SUM4(yv[0],yv[1],yv[2],yv[3])) ;
   dzv = Real_t(0.25)*(SUM4(zv[4],zv[5],zv[6],zv[7]) - SUM4(zv[0],zv[1],zv[2],zv[3])) ;

   *delv_zeta = ax*dxv + ay*dyv + az*dzv ;

   /* find delxi and delvi ( j cross k ) */

   ax = dyj*dzk - dzj*dyk ;
   ay = dzj*dxk - dxj*dzk ;
   az = dxj*dyk - dyj*dxk ;

   *delx_xi = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

   ax *= norm ;
   ay *= norm ;
   az *= norm ;

   dxv = Real_t(0.25)*(SUM4(xv[1],xv[2],xv[6],xv[5]) - SUM4(xv[0],xv[3],xv[7],xv[4])) ;
   dyv = Real_t(0.25)*(SUM4(yv[1],yv[2],yv[6],yv[5]) - SUM4(yv[0],yv[3],yv[7],yv[4])) ;
   dzv = Real_t(0.25)*(SUM4(zv[1],zv[2],zv[6],zv[5]) - SUM4(zv[0],zv[3],zv[7],zv[4])) ;

   *delv_xi = ax*dxv + ay*dyv + az*dzv ;

   /* find delxj and delvj ( k cross i ) */

   ax = dyk*dzi - dzk*dyi ;
   ay = dzk*dxi - dxk*dzi ;
   az = dxk*dyi - dyk*dxi ;

   *delx_eta = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

   ax *= norm ;
   ay *= norm ;
   az *= norm ;

   dxv = Real_t(-0.25)*(SUM4(xv[0],xv[1],xv[5],xv[4]) - SUM4(xv[3],xv[2],xv[6],xv[7])) ;
   dyv = Real_t(-0.25)*(SUM4(yv[0],yv[1],yv[5],yv[4]) - SUM4(yv[3],yv[2],yv[6],yv[7])) ;
   dzv = Real_t(-0.25)*(SUM4(zv[0],zv[1],zv[5],zv[4]) - SUM4(zv[3],zv[2],zv[6],zv[7])) ;

   *delv_eta = ax*dxv + ay*dyv + az*dzv ;
#undef SUM4
}


__device__
static
__forceinline__
void CalcElemVelocityGradient( const Real_t* const xvel,
                                const Real_t* const yvel,
                                const Real_t* const zvel,
                                const Real_t b[][8],
                                const Real_t detJ,
                                Real_t* const d )
{
  const Real_t inv_detJ = Real_t(1.0) / detJ ;
  Real_t dyddx, dxddy, dzddx, dxddz, dzddy, dyddz;
  const Real_t* const pfx = b[0];
  const Real_t* const pfy = b[1];
  const Real_t* const pfz = b[2];

  d[0] = inv_detJ * ( pfx[0] * (xvel[0]-xvel[6])
                     + pfx[1] * (xvel[1]-xvel[7])
                     + pfx[2] * (xvel[2]-xvel[4])
                     + pfx[3] * (xvel[3]-xvel[5]) );

  d[1] = inv_detJ * ( pfy[0] * (yvel[0]-yvel[6])
                     + pfy[1] * (yvel[1]-yvel[7])
                     + pfy[2] * (yvel[2]-yvel[4])
                     + pfy[3] * (yvel[3]-yvel[5]) );

  d[2] = inv_detJ * ( pfz[0] * (zvel[0]-zvel[6])
                     + pfz[1] * (zvel[1]-zvel[7])
                     + pfz[2] * (zvel[2]-zvel[4])
                     + pfz[3] * (zvel[3]-zvel[5]) );

  dyddx  = inv_detJ * ( pfx[0] * (yvel[0]-yvel[6])
                      + pfx[1] * (yvel[1]-yvel[7])
                      + pfx[2] * (yvel[2]-yvel[4])
                      + pfx[3] * (yvel[3]-yvel[5]) );

  dxddy  = inv_detJ * ( pfy[0] * (xvel[0]-xvel[6])
                      + pfy[1] * (xvel[1]-xvel[7])
                      + pfy[2] * (xvel[2]-xvel[4])
                      + pfy[3] * (xvel[3]-xvel[5]) );

  dzddx  = inv_detJ * ( pfx[0] * (zvel[0]-zvel[6])
                      + pfx[1] * (zvel[1]-zvel[7])
                      + pfx[2] * (zvel[2]-zvel[4])
                      + pfx[3] * (zvel[3]-zvel[5]) );

  dxddz  = inv_detJ * ( pfz[0] * (xvel[0]-xvel[6])
                      + pfz[1] * (xvel[1]-xvel[7])
                      + pfz[2] * (xvel[2]-xvel[4])
                      + pfz[3] * (xvel[3]-xvel[5]) );

  dzddy  = inv_detJ * ( pfy[0] * (zvel[0]-zvel[6])
                      + pfy[1] * (zvel[1]-zvel[7])
                      + pfy[2] * (zvel[2]-zvel[4])
                      + pfy[3] * (zvel[3]-zvel[5]) );

  dyddz  = inv_detJ * ( pfz[0] * (yvel[0]-yvel[6])
                      + pfz[1] * (yvel[1]-yvel[7])
                      + pfz[2] * (yvel[2]-yvel[4])
                      + pfz[3] * (yvel[3]-yvel[5]) );
  d[5]  = Real_t( .5) * ( dxddy + dyddx );
  d[4]  = Real_t( .5) * ( dxddz + dzddx );
  d[3]  = Real_t( .5) * ( dzddy + dyddz );
}


__device__
static inline
Real_t AreaFace( const Real_t x0, const Real_t x1,
                 const Real_t x2, const Real_t x3,
                 const Real_t y0, const Real_t y1,
                 const Real_t y2, const Real_t y3,
                 const Real_t z0, const Real_t z1,
                 const Real_t z2, const Real_t z3)
{
   Real_t fx = (x2 - x0) - (x3 - x1);
   Real_t fy = (y2 - y0) - (y3 - y1);
   Real_t fz = (z2 - z0) - (z3 - z1);
   Real_t gx = (x2 - x0) + (x3 - x1);
   Real_t gy = (y2 - y0) + (y3 - y1);
   Real_t gz = (z2 - z0) + (z3 - z1);
   Real_t area =
      (fx * fx + fy * fy + fz * fz) *
      (gx * gx + gy * gy + gz * gz) -
      (fx * gx + fy * gy + fz * gz) *
      (fx * gx + fy * gy + fz * gz);
   return area ;
}


__device__
static inline
Real_t CalcElemCharacteristicLength( const Real_t x[8],
                                     const Real_t y[8],
                                     const Real_t z[8],
                                     const Real_t volume)
{
   Real_t a, charLength = Real_t(0.0);

   a = AreaFace(x[0],x[1],x[2],x[3],
                y[0],y[1],y[2],y[3],
                z[0],z[1],z[2],z[3]) ; // 38
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[4],x[5],x[6],x[7],
                y[4],y[5],y[6],y[7],
                z[4],z[5],z[6],z[7]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[0],x[1],x[5],x[4],
                y[0],y[1],y[5],y[4],
                z[0],z[1],z[5],z[4]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[1],x[2],x[6],x[5],
                y[1],y[2],y[6],y[5],
                z[1],z[2],z[6],z[5]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[2],x[3],x[7],x[6],
                y[2],y[3],y[7],y[6],
                z[2],z[3],z[7],z[6]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[3],x[0],x[4],x[7],
                y[3],y[0],y[4],y[7],
                z[3],z[0],z[4],z[7]) ;
   charLength = FMAX(a,charLength) ;

   charLength = Real_t(4.0) * volume / SQRT(charLength);

   return charLength;
}

extern "C" __global__
#ifdef DOUBLE_PRECISION
__launch_bounds__(64,8) // 64-bit
#else
__launch_bounds__(64,16) // 32-bit
#endif
  void CalcKinematicsAndMonotonicQGradient_kernel(occaKernelInfoArg,
    Index_t numElem, Index_t padded_numElem, const Real_t dt,
    const Index_t* __restrict__ nodelist, const Real_t* __restrict__ volo, const Real_t* __restrict__ v,
						  Real_t * __restrict__ x, //TextureObj<Real_t> x,
						  Real_t * __restrict__ y, //TextureObj<Real_t> y,
						  Real_t * __restrict__ z, //TextureObj<Real_t> z,
						  Real_t * __restrict__ xd,//TextureObj<Real_t> xd,
						  Real_t * __restrict__ yd,//TextureObj<Real_t> yd,
						  Real_t * __restrict__ zd,//TextureObj<Real_t> zd,
    Real_t* __restrict__ vnew,
    Real_t* __restrict__ delv,
    Real_t* __restrict__ arealg,
    Real_t* __restrict__ dxx,
    Real_t* __restrict__ dyy,
    Real_t* __restrict__ dzz,
    Real_t* __restrict__ vdov,
    Real_t* __restrict__ delx_zeta,
    Real_t* __restrict__ delv_zeta,
    Real_t* __restrict__ delx_xi,
    Real_t* __restrict__ delv_xi,
    Real_t* __restrict__ delx_eta,
    Real_t* __restrict__ delv_eta,
    Index_t* __restrict__ bad_vol,
    const Index_t num_threads
    )
{

  Real_t B[3][8] ; /** shape function derivatives */
  Index_t nodes[8] ;
  Real_t x_local[8] ;
  Real_t y_local[8] ;
  Real_t z_local[8] ;
  Real_t xd_local[8] ;
  Real_t yd_local[8] ;
  Real_t zd_local[8] ;
  Real_t D[6];

  int k=blockDim.x*blockIdx.x+threadIdx.x;

  if ( k < num_threads) {

    Real_t volume ;
    Real_t relativeVolume ;

    // get nodal coordinates from global arrays and copy into local arrays.
    #pragma unroll
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = nodelist[k+lnode*padded_numElem];
      nodes[lnode] = gnode;
      x_local[lnode] = x[gnode];
      y_local[lnode] = y[gnode];
      z_local[lnode] = z[gnode];
    }

    // volume calculations
    volume = CalcElemVolume(x_local, y_local, z_local );

    relativeVolume = volume / volo[k] ;
    vnew[k] = relativeVolume ;

    delv[k] = relativeVolume - v[k] ;
    // set characteristic length
    arealg[k] = CalcElemCharacteristicLength(x_local,y_local,z_local,volume);

    // get nodal velocities from global array and copy into local arrays.
    #pragma unroll
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = nodes[lnode];
      xd_local[lnode] = xd[gnode];
      yd_local[lnode] = yd[gnode];
      zd_local[lnode] = zd[gnode];
    }

    Real_t dt2 = Real_t(0.5) * dt;

    #pragma unroll
    for ( Index_t j=0 ; j<8 ; ++j )
    {
       x_local[j] -= dt2 * xd_local[j];
       y_local[j] -= dt2 * yd_local[j];
       z_local[j] -= dt2 * zd_local[j];
    }

    Real_t detJ;

    CalcElemShapeFunctionDerivatives(x_local,y_local,z_local,B,&detJ );

    CalcElemVelocityGradient(xd_local,yd_local,zd_local,B,detJ,D);

    // ------------------------
    // CALC LAGRANGE ELEM 2
    // ------------------------

    // calc strain rate and apply as constraint (only done in FB element)
    Real_t vdovNew = D[0] + D[1] + D[2];
    Real_t vdovthird = vdovNew/Real_t(3.0) ;

    // make the rate of deformation tensor deviatoric
    vdov[k] = vdovNew ;
    dxx[k] = D[0] - vdovthird ;
    dyy[k] = D[1] - vdovthird ;
    dzz[k] = D[2] - vdovthird ;

    // ------------------------
    // CALC MONOTONIC Q GRADIENT
    // ------------------------
    Real_t vol = volo[k]*vnew[k];

   // Undo x_local update
    #pragma unroll
    for ( Index_t j=0 ; j<8 ; ++j ) {
       x_local[j] += dt2 * xd_local[j];
       y_local[j] += dt2 * yd_local[j];
       z_local[j] += dt2 * zd_local[j];
    }

   CalcMonoGradient(x_local,y_local,z_local,xd_local,yd_local,zd_local,
                          vol,
                          &delx_zeta[k],&delv_zeta[k],&delx_xi[k],
                          &delv_xi[k], &delx_eta[k], &delv_eta[k]);

  //Check for bad volume
  if (relativeVolume < 0)
    *bad_vol = k;
  }
}
