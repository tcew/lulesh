#include "hip/hip_runtime.h"
#include<kernelDefines.cu>

extern "C" __global__
void CalcPositionAndVelocityForNodes_kernel(occaKernelInfoArg, int numNode,
    const Real_t deltatime,
    const Real_t u_cut,
    Real_t* __restrict__ x,  Real_t* __restrict__ y,  Real_t* __restrict__ z,
    Real_t* __restrict__ xd, Real_t* __restrict__ yd, Real_t* __restrict__ zd,
    const Real_t* __restrict__ xdd, const Real_t* __restrict__ ydd, const Real_t* __restrict__ zdd)
{
    int i=blockDim.x*blockIdx.x+threadIdx.x;
    if (i < numNode)
    {
      Real_t xdtmp, ydtmp, zdtmp, dt;
      dt = deltatime;

      xdtmp = xd[i] + xdd[i] * dt ;
      ydtmp = yd[i] + ydd[i] * dt ;
      zdtmp = zd[i] + zdd[i] * dt ;

      if( FABS(xdtmp) < u_cut ) xdtmp = 0.0;
      if( FABS(ydtmp) < u_cut ) ydtmp = 0.0;
      if( FABS(zdtmp) < u_cut ) zdtmp = 0.0;

      x[i] += xdtmp * dt;
      y[i] += ydtmp * dt;
      z[i] += zdtmp * dt;

      xd[i] = xdtmp;
      yd[i] = ydtmp;
      zd[i] = zdtmp;
    }
}


/* #include<kernelDefines.occa> */

/* occaKernel void */
/* CalcPositionAndVelocityForNodes_kernel(occaKernelInfoArg, */
/* 				       const int occaVariable numNode, */
/* 				       const Real_t occaVariable deltatime, */
/* 				       const Real_t occaVariable u_cut, */
/* 				       occaPointer Real_t* occaRestrict x, */
/* 				       occaPointer Real_t* occaRestrict y, */
/* 				       occaPointer Real_t* occaRestrict z, */
/* 				       occaPointer Real_t* occaRestrict xd, */
/* 				       occaPointer Real_t* occaRestrict yd, */
/* 				       occaPointer Real_t* occaRestrict zd, */
/* 				       const occaPointer Real_t* occaRestrict xdd, */
/* 				       const occaPointer Real_t* occaRestrict ydd, */
/* 				       const occaPointer Real_t* occaRestrict zdd){ */

/*   occaGlobalFor0{ */
/*     // int i=blockDim.x*blockIdx.x+threadIdx.x; */
/*     const int i = occaGlobalId0; */
/*     if (i < numNode){ */

/*       Real_t xdtmp, ydtmp, zdtmp, dt; */
/*       dt = deltatime; */

/*       xdtmp = xd[i] + xdd[i] * dt ; */
/*       ydtmp = yd[i] + ydd[i] * dt ; */
/*       zdtmp = zd[i] + zdd[i] * dt ; */

/*       if( FABS(xdtmp) < u_cut ) xdtmp = 0.0; */
/*       if( FABS(ydtmp) < u_cut ) ydtmp = 0.0; */
/*       if( FABS(zdtmp) < u_cut ) zdtmp = 0.0; */

/*       x[i] += xdtmp * dt; */
/*       y[i] += ydtmp * dt; */
/*       z[i] += zdtmp * dt; */

/*       xd[i] = xdtmp; */
/*       yd[i] = ydtmp; */
/*       zd[i] = zdtmp; */
/*     } */
/*   } */
/* } */
